#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <sys/time.h>
#include "cuPrintf.cu"
#include "cuPrintf.cuh"

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}



#define PRINT_TIME         1
#define FILE_NAME "test2.txt"
#define MAX_THREAD 1024

#define IMUL(a, b) __mul24(a, b)


void LoadInput(char file[], 
  int* num_painting, int* num_bags,
  int** weights, int** values){
FILE *infile; 
char filename[64];
int i, temp_int;
snprintf(filename, sizeof(filename), "%s", file);

// Open input txt file with error check 
infile = fopen (filename, "r"); 
if (infile == NULL) {
fprintf(stderr, "\nError opening file\n"); 
exit (1); 
} 

// Read integers

// load number of painting in the first line
fscanf (infile, "%d", &temp_int);
*num_painting = temp_int;

// load number of bags in the first line
fscanf (infile, "%d", num_bags);


// prepare memory for weights and values array
*weights = (int *) malloc((*num_painting)*sizeof(int));
*values = (int *) malloc((*num_painting)*sizeof(int));

// load weights
for(i=0; i<(*num_painting); i++){
  fscanf (infile, "%d", &temp_int);
  (*weights)[i] = temp_int;
  }

  // load values
  for(i=0; i<(*num_painting); i++){
  fscanf (infile, "%d", &temp_int);
  (*values)[i] = temp_int;
  }
  fclose (infile); 

  printf("number of bags = %d\n", *num_bags);
  printf("number of paintings = %d\n", *num_painting);

}

void Worker(int n, int b, int* weight, int* value, int* result){
  int i, j;
  int* tmp1 = (int *) malloc(b*sizeof(int));
  int* tmp2 = (int *) malloc(b*sizeof(int));
  int* tmp3;
  for(j=0; j<b; j++){
    if (weight[0] > j) {
      tmp1[j] = 0;
    } 
    else {
      tmp1[j] = value[0];
    }
  }

  for(i = 1; i<n; i++){
    //printf("i = %d\n",i);
    for(j=0; j<b; j++){
      //printf("j = %d\n",j);
      if (j < weight[i] || tmp1[j] >= tmp1[j-weight[i]] + value[i]){
        //printf("er\n");
        tmp2[j] = tmp1[j];
      }
      else{
        tmp2[j] = tmp1[j-weight[i]] + value[i];
      }
    }
    tmp3 = tmp1;
    tmp1 = tmp2;
    tmp2 = tmp3;
  }
  for(j=0; j<b; j++){
    result[j] = tmp1[j];
  }
}




void print_result(int num_bags, int* result){
  int j;
    for(j=0; j < num_bags;j++){
      printf("%d ",result[j]);
    }
    printf("\n");
}


__global__ void kernel_initial (int col, int* weight, int* v, int* this_col) {
	
  int row = threadIdx.y+MAX_THREAD*blockIdx.y;
  if (weight[0] > row) {
    this_col[row] = 0;
  } 
  else {
    this_col[row] = v[0];
  }
}

__global__ void kernel (int col, int* w, int* v, int* this_col, int* last_col) {
	
  int row = threadIdx.y+MAX_THREAD*blockIdx.y;
  if (row < w[col] || last_col[row] >= last_col[row-w[col]] + v[col]){
    this_col[row] = last_col[row];
  }
  else{
    this_col[row] = last_col[row-w[col]] + v[col];
  }
}



int main(int argc, char **argv){
  //num of paintings and bags
  long int num_painting,num_bags, temp_long_int;

  //some temp variables
  int i, temp_int;
  
  //store cpu time
  struct timeval end, begin;

  // Arrays on the host memory
  int* weights;
  int* values;
  int* results;
  int* results_gold;
  // load input from txt
  
  FILE *infile; 
  char filename[64] = FILE_NAME;

  // Open input txt file with error check 
  infile = fopen (filename, "r"); 
  if (infile == NULL) {
    fprintf(stderr, "\nError opening file\n"); 
    exit (1); 
  } 

  // Read integers
  // load number of painting in the first line
  fscanf (infile, "%ld", &temp_long_int);
  num_painting = temp_long_int;

 

  // prepare memory for weights and values array
  weights = (int *) malloc(num_painting*sizeof(int));
  values = (int *) malloc(num_painting*sizeof(int));

  // load weights
  	for(i=0; i<num_painting; i++){
    	fscanf (infile, "%d", &temp_int);
    	weights[i] = temp_int;
    }
    
    // load number of bags in the first line
  	fscanf (infile, "%ld", &temp_long_int);
	  num_bags = temp_long_int;

    // load values
    for(i=0; i<num_painting; i++){
    	fscanf (infile, "%d", &temp_int);
    	values[i] = temp_int;
    }
    
    fclose (infile); 
    printf("number of bags = %ld\n", num_bags);
    printf("number of paintings = %ld\n", num_painting);


   // Allocate arrays on host memory
   results = (int *) malloc(num_bags * sizeof(int));
   results_gold = (int *) malloc(num_bags * sizeof(int));
  
  // GPU Timing variables
  hipEvent_t start, stop;
  float elapsed_gpu;

  // Arrays on GPU global memoryc
  int *gpu_weights;
  int *gpu_values;
  int *col_1;
  int *col_2;

  // Select GPU
  CUDA_SAFE_CALL(hipSetDevice(0));

  // Allocate GPU memory
  size_t allocSize_1 = num_painting * sizeof(int);
  size_t allocSize_2 = num_bags * sizeof(int);
  CUDA_SAFE_CALL(hipMalloc((void **)&gpu_weights, allocSize_1));
  CUDA_SAFE_CALL(hipMalloc((void **)&gpu_values, allocSize_1));
  CUDA_SAFE_CALL(hipMalloc((void **)&col_1, allocSize_2));
  CUDA_SAFE_CALL(hipMalloc((void **)&col_2, allocSize_2));

  printf("Allocate done\n\n");
  

#if PRINT_TIME
  // Create the cuda events
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // Record event on the default stream
  
#endif

  // Transfer the arrays to the GPU memory
  CUDA_SAFE_CALL(hipMemcpy(gpu_weights, weights, allocSize_1, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(gpu_values, values, allocSize_1, hipMemcpyHostToDevice));

	// init cuPrint
	cudaPrintfInit ();
	
	dim3 dimGrid(1,num_bags/MAX_THREAD,1);
	dim3 dimBlock(1,MAX_THREAD,1);

  hipEventRecord(start, 0);
  // Launch the kernel
  kernel_initial<<<dimGrid, dimBlock>>>(i,  gpu_weights, gpu_values, col_1);
  for(i=1;i<num_painting;i++){
    if(i%2)
      kernel<<<dimGrid, dimBlock>>>(i,  gpu_weights, gpu_values, col_2, col_1);
    else
      kernel<<<dimGrid, dimBlock>>>(i,  gpu_weights, gpu_values, col_1, col_2);
  }

  
  hipEventRecord(stop,0);
  // end of cuPrint
  cudaPrintfDisplay (stdout, true);
	cudaPrintfEnd ();

  // Check for errors during launch
  CUDA_SAFE_CALL(hipPeekAtLastError());

  // Transfer the results back to the host
  if(num_painting%2){
    CUDA_SAFE_CALL(hipMemcpy(results, col_1, allocSize_2, hipMemcpyDeviceToHost));
  }
  else{
    CUDA_SAFE_CALL(hipMemcpy(results, col_2, allocSize_2, hipMemcpyDeviceToHost));
  }
#if PRINT_TIME
  // Stop and destroy the timer
  
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_gpu, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
#endif

  printf("gpu_result:\n");
  print_result(num_bags, results);
  
  
  // Compute the results on the host

  gettimeofday(&begin, NULL);
  Worker(num_painting, num_bags, weights, values, results_gold);
  gettimeofday(&end, NULL);

  
  printf("cpu_result:\n");
	print_result(num_bags, results_gold);

  printf("cpu time =  %lu us\n", (end.tv_sec - begin.tv_sec) * 1000000 + end.tv_usec - begin.tv_usec);
  printf("\nGPU time: %f (usec)\n", elapsed_gpu * 1000);
  // Free-up device and host memory
  CUDA_SAFE_CALL(hipFree(gpu_weights));
  CUDA_SAFE_CALL(hipFree(gpu_values));
  CUDA_SAFE_CALL(hipFree(col_1));
  CUDA_SAFE_CALL(hipFree(col_2));

  free(weights);
  free(values);
  free(results);
  free(results_gold);

  return 0;
}
